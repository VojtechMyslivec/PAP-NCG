#include "hip/hip_runtime.h"
/** dijkstra.cu
 *
 * Autori:      Vojtech Myslivec <vojtech.myslivec@fit.cvut.cz>,  FIT CVUT v Praze
 *              Zdenek  Novy     <novyzde3@fit.cvut.cz>,          FIT CVUT v Praze
 *              
 * Datum:       unor-duben 2015
 *
 * Popis:       Semestralni prace z predmetu MI-PAP:
 *              Hledani nejkratsich cest v grafu 
 *                 paralelni implementace na CUDA
 *                 algoritmus Dijkstra
 *
 *
 */

#include "dijkstra.cuh"
#include "cDijkstra.cuh"
#include "funkceSpolecne.cuh"

// TODO smazat
//#include <iostream>
//#include <fstream>
//#include <iomanip>
//#include <cstring>


using namespace std;

void dijkstraObjektInit( unsigned ** devGraf, unsigned pocetUzlu, unsigned idUzlu, cDijkstra *& devDijkstra ) {
    // vytvori objekt na host, aby ho zkopiroval na device ----------
    cDijkstra * hostDijkstra = new cDijkstra( pocetUzlu, idUzlu ); 

    hipMalloc( &devDijkstra, sizeof(*devDijkstra) );
    hipMemcpy( devDijkstra, hostDijkstra, sizeof(*devDijkstra), hipMemcpyHostToDevice );

    delete hostDijkstra;

    // alokace vnitrnich poli ---------------------------------------
    unsigned * devVzdalenost;
    bool     * devUzavreny;
    HANDLE_ERROR( 
       hipMalloc( 
                   &devVzdalenost,
                   pocetUzlu*sizeof(*devVzdalenost)
                 )
    );
    HANDLE_ERROR( 
       hipMalloc( 
                   &devUzavreny,
                   pocetUzlu*sizeof(*devUzavreny)
                 )
    );

    // zkopirovani pointeru do instance -----------------------------
    HANDLE_ERROR( 
       hipMemcpy( 
                   &(devDijkstra->vzdalenost),
                   &(devVzdalenost),
                   sizeof(devVzdalenost),
                   hipMemcpyHostToDevice
                 )
    );
    HANDLE_ERROR( 
       hipMemcpy( 
                   &(devDijkstra->uzavreny),
                   &(devUzavreny),
                   sizeof(devUzavreny),
                   hipMemcpyHostToDevice
                 )
    );

    // zkopruje ukazatel na pseudo-staticky graf --------------------
    HANDLE_ERROR( 
       hipMemcpy( 
                   &(devDijkstra->graf),
                   &(devGraf),
                   sizeof(devGraf),
                   hipMemcpyHostToDevice
                 )
    );
}

void dijkstraInicializaceNaGPU( unsigned ** devGraf, unsigned pocetUzlu, cDijkstra **& devDijkstra ) {
    cDijkstra ** hostDevDijkstra = new cDijkstra * [pocetUzlu];

    // alokace objektu na GPU ---------------------------------------
    for ( unsigned idUzlu = 0 ; idUzlu < pocetUzlu ; idUzlu++ ) {
        dijkstraObjektInit( devGraf, pocetUzlu, idUzlu, hostDevDijkstra[idUzlu] );
    }
    // alokace a zkopirovani pole ukazatelu na GPU ------------------
    HANDLE_ERROR( 
       hipMalloc( 
                   &devDijkstra,
                   pocetUzlu*sizeof(*devDijkstra)
                 )
    );
    HANDLE_ERROR( 
       hipMemcpy( 
                   devDijkstra,
                   hostDevDijkstra,
                   pocetUzlu*sizeof(*devDijkstra), 
                   hipMemcpyHostToDevice 
                 )
    );

    delete [] hostDevDijkstra;
}

void inicializaceNtoN( unsigned **  graf, unsigned pocetUzlu, 
                       unsigned **& vzdalenostM,
                       unsigned **& devGraf, cDijkstra **& devDijkstra
                     ) {
    // inicializace matic vysledku -------------------------
    maticeInicializaceNaCPU( vzdalenostM, pocetUzlu );

    // zkopirovani grafu na GPU ----------------------------
    maticeInicializaceNaGPU( graf, pocetUzlu, devGraf );

    // inicializace objektu na GPU -------------------------
    dijkstraInicializaceNaGPU( devGraf, pocetUzlu, devDijkstra );

    // dalsi nastaveni pro GPU -----------------------------
#ifdef CACHE
    hipFuncSetCacheConfig(reinterpret_cast<const void*>( wrapperProGPU), hipFuncCachePreferL1 );
    //hipDeviceSetCacheConfig( hipFuncCachePreferL1 );
#endif // CACHE
}

void dijkstraObjektUklid( cDijkstra *& devDijkstra ) {
    unsigned * devVzdalenost;
    bool     * devUzavreny;
    // zkopirovani pointeru z [instance na GPU] ---------------------
    HANDLE_ERROR( 
            hipMemcpy( 
                &(devVzdalenost),
                &(devDijkstra->vzdalenost),
                sizeof(devVzdalenost),
                hipMemcpyDeviceToHost
                )
            );
    HANDLE_ERROR( 
            hipMemcpy( 
                &(devUzavreny),
                &(devDijkstra->uzavreny),
                sizeof(devUzavreny),
                hipMemcpyDeviceToHost
                )
            );
    // uvolneni pameti vnitrnich poli instance ----------------------
    HANDLE_ERROR( 
            hipFree( devVzdalenost )
            );
    HANDLE_ERROR( 
            hipFree( devUzavreny )
            );

    // uvolni objekt z pameti na GPU --------------------------------
    hipFree( devDijkstra );
    devDijkstra = NULL;
}

void dijkstraUklidNaGPU( cDijkstra **& devDijkstra, unsigned pocetUzlu ) {
    // zkopirovani pole [ukazatelu do device] ---------------------------
    cDijkstra ** hostDevDijkstra = new cDijkstra * [pocetUzlu];
    HANDLE_ERROR( 
       hipMemcpy( 
                   hostDevDijkstra,
                   devDijkstra,
                   pocetUzlu*sizeof(*devDijkstra), 
                   hipMemcpyDeviceToHost 
                 )
    );

    // uvolneni objektu z pameti GPU ---------------------------------
    for ( unsigned idUzlu = 0 ; idUzlu < pocetUzlu ; idUzlu++ ) {
        dijkstraObjektUklid( hostDevDijkstra[idUzlu] );
    }
    // uvolneni pole ukazatelu na objekty na GPU --------------------
    HANDLE_ERROR( 
            hipFree( devDijkstra )
            );
    devDijkstra = NULL;
    delete [] hostDevDijkstra;
}

void uklidNtoN( unsigned  ** vzdalenostM, 
                unsigned  ** devGraf,
                cDijkstra ** devDijkstra,
                unsigned     pocetUzlu 
              ) {
    maticeUklidNaCPU(   vzdalenostM, pocetUzlu );
    dijkstraUklidNaGPU( devDijkstra, pocetUzlu );
    maticeUklidNaGPU(       devGraf, pocetUzlu );
}

void zkopirujDataZGPU( unsigned ** vzdalenostM, cDijkstra ** devDijkstra, unsigned pocetUzlu ) {
    // zkopirovani pole [ukazatelu do device] ---------------------------
    cDijkstra ** hostDevDijkstra = new cDijkstra * [pocetUzlu];
    HANDLE_ERROR( 
       hipMemcpy( 
                   hostDevDijkstra,
                   devDijkstra,
                   pocetUzlu*sizeof(*devDijkstra), 
                   hipMemcpyDeviceToHost 
                 )
    );

    unsigned * devHodnoty;
    for ( unsigned i = 0 ; i < pocetUzlu ; i++ ) {
        // zkopiruje hodnotu ukazatele z [tridy na device] ----------
        HANDLE_ERROR( 
            hipMemcpy( 
                        &(devHodnoty), 
                        &(hostDevDijkstra[i]->vzdalenost), 
                        //&(devDijkstra[i]->vzdalenost), 
                        sizeof(devHodnoty), 
                        hipMemcpyDeviceToHost
                      )
        );

        // zkopiruje data z device do matice vzdalenosti ------------
        HANDLE_ERROR( 
           hipMemcpy(
                       vzdalenostM[i],
                       devHodnoty,
                       pocetUzlu*sizeof(*devHodnoty),
                       hipMemcpyDeviceToHost 
                     )
        );
    }
    
    delete [] hostDevDijkstra;
}

__global__ void wrapperProGPU( cDijkstra ** devDijkstra, unsigned pocetUzlu, unsigned pocetVlakenVBloku ) {
    int blok   =  blockIdx.x;
    int vlakno = threadIdx.x;
    int i      = pocetVlakenVBloku * blok + vlakno;

#ifdef DEBUG
    printf( "thread id = %d, b = %d, v = %d\n", i, blok, vlakno );
#endif // DEBUG

    if ( i < pocetUzlu ) {
        devDijkstra[i]->devInicializujHodnoty();
        devDijkstra[i]->devSpustVypocet();
    }
    return;
}

bool dijkstraNtoN( unsigned ** graf, unsigned pocetUzlu, unsigned pocetWarpu ) {
    unsigned  ** devGraf;
    unsigned  ** vzdalenostM; 
    cDijkstra ** devDijkstra;
    // pocet vlaken v bloku -- minimalne pocet uzlu
    int vlaken = MIN( pocetWarpu * CUDA_WARP_VELIKOST, pocetUzlu );
    // horni cast pocetUzlu/vlaken
    int bloku  = ( pocetUzlu + vlaken - 1 ) / vlaken;
#ifdef MERENI
    // udalosti pro mereni casu vypoctu
    hipEvent_t eStart, eZapsano, eVypocteno, eKonec;
    float       tVypocet, tCelkem;

    HANDLE_ERROR(   hipEventCreate(     &eStart )  );
    HANDLE_ERROR(   hipEventCreate(   &eZapsano )  );
    HANDLE_ERROR(   hipEventCreate( &eVypocteno )  );
    HANDLE_ERROR(   hipEventCreate(     &eKonec )  );

    HANDLE_ERROR(   hipEventRecord(      eStart )  );
    // event synchronize, aby se vsechny operace dokoncily a mereni
    // probehlo v poradku
    HANDLE_ERROR(   hipEventSynchronize( eStart ) );
#endif // MERENI

    // inicializace a kopirovani dat na GPU --------------------------
    inicializaceNtoN( graf, pocetUzlu, vzdalenostM, devGraf, devDijkstra );

#ifdef MERENI
    HANDLE_ERROR(   hipEventRecord(    eZapsano )  );
    HANDLE_ERROR( hipEventSynchronize( eZapsano )  );
#endif // MERENI

    // vypocet na GPU ------------------------------------------------
    wrapperProGPU<<<bloku,vlaken>>>( devDijkstra, pocetUzlu, vlaken ) ;
    HANDLE_ERROR(   hipDeviceSynchronize( )        );

#ifdef MERENI
    HANDLE_ERROR(   hipEventRecord(  eVypocteno )  );
    HANDLE_ERROR( hipEventSynchronize( eVypocteno ) );
#endif // MERENI

    // kopirovani dat z GPU ------------------------------------------
    zkopirujDataZGPU( vzdalenostM, devDijkstra, pocetUzlu );

#ifdef VYPIS
    // vypis vysledku ------------------------------------------------
    vypisGrafu( cout, vzdalenostM, pocetUzlu );
#endif // VYPIS

    // uvolneni pameti na CPU i GPU ----------------------------------
    uklidNtoN( vzdalenostM, devGraf, devDijkstra, pocetUzlu );

#ifdef MERENI
    HANDLE_ERROR(   hipEventRecord(      eKonec )  );
    HANDLE_ERROR(   hipEventSynchronize( eKonec )  );

    HANDLE_ERROR(   hipEventElapsedTime( &tVypocet, eZapsano, eVypocteno )  );
    HANDLE_ERROR(   hipEventElapsedTime(  &tCelkem,   eStart,     eKonec )  );

    tCelkem  /= 1000;
    tVypocet /= 1000;
    cerr << pocetUzlu << '	' << bloku   << '	' << vlaken << '	'
         << tVypocet  << '	' << tCelkem << '	' << endl;


    HANDLE_ERROR(  hipEventDestroy(     eKonec )  );
    HANDLE_ERROR(  hipEventDestroy( eVypocteno )  );
    HANDLE_ERROR(  hipEventDestroy(   eZapsano )  );
    HANDLE_ERROR(  hipEventDestroy(     eStart )  );
#endif // MERENI

    return true;
}

