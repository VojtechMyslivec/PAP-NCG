#include "hip/hip_runtime.h"
/** dijkstra.cu
 *
 * Autori:      Vojtech Myslivec <vojtech.myslivec@fit.cvut.cz>,  FIT CVUT v Praze
 *              Zdenek  Novy     <novyzde3@fit.cvut.cz>,          FIT CVUT v Praze
 *              
 * Datum:       unor-duben 2015
 *
 * Popis:       Semestralni prace z predmetu MI-PAP:
 *              Hledani nejkratsich cest v grafu 
 *                 paralelni implementace na CUDA
 *                 algoritmus Dijkstra
 *
 *
 */

#include "dijkstra.cuh"
#include "cDijkstra.cuh"
#include "funkceSpolecne.cuh"

// TODO smazat
//#include <iostream>
//#include <fstream>
//#include <iomanip>
//#include <cstring>


using namespace std;

void dijkstraObjektInit( unsigned ** devGraf, unsigned pocetUzlu, unsigned idUzlu, cDijkstra *& devDijkstra ) {
    // vytvori objekt na host, aby ho zkopiroval na device ----------
    cDijkstra * hostDijkstra = new cDijkstra( pocetUzlu, idUzlu ); 

    hipMalloc( &devDijkstra, sizeof(*devDijkstra) );
    hipMemcpy( devDijkstra, hostDijkstra, sizeof(*devDijkstra), hipMemcpyHostToDevice );

    delete hostDijkstra;

    // alokace vnitrnich poli ---------------------------------------
    unsigned * devVzdalenost;
    bool     * devUzavreny;
    HANDLE_ERROR( 
       hipMalloc( 
                   &devVzdalenost,
                   pocetUzlu*sizeof(*devVzdalenost)
                 )
    );
    HANDLE_ERROR( 
       hipMalloc( 
                   &devUzavreny,
                   pocetUzlu*sizeof(*devUzavreny)
                 )
    );

    // zkopirovani pointeru do instance -----------------------------
    HANDLE_ERROR( 
       hipMemcpy( 
                   &(devDijkstra->vzdalenost),
                   &(devVzdalenost),
                   sizeof(devVzdalenost),
                   hipMemcpyHostToDevice
                 )
    );
    HANDLE_ERROR( 
       hipMemcpy( 
                   &(devDijkstra->uzavreny),
                   &(devUzavreny),
                   sizeof(devUzavreny),
                   hipMemcpyHostToDevice
                 )
    );

    // zkopruje ukazatel na pseudo-staticky graf --------------------
    HANDLE_ERROR( 
       hipMemcpy( 
                   &(devDijkstra->graf),
                   &(devGraf),
                   sizeof(devGraf),
                   hipMemcpyHostToDevice
                 )
    );
}

void dijkstraInicializaceNaGPU( unsigned ** devGraf, unsigned pocetUzlu, cDijkstra **& devDijkstra ) {
    cDijkstra ** hostDevDijkstra = new cDijkstra * [pocetUzlu];

    // alokace objektu na GPU ---------------------------------------
    for ( unsigned idUzlu = 0 ; idUzlu < pocetUzlu ; idUzlu++ ) {
        dijkstraObjektInit( devGraf, pocetUzlu, idUzlu, hostDevDijkstra[idUzlu] );
    }
    // alokace a zkopirovani pole ukazatelu na GPU ------------------
    HANDLE_ERROR( 
       hipMalloc( 
                   &devDijkstra,
                   pocetUzlu*sizeof(*devDijkstra)
                 )
    );
    HANDLE_ERROR( 
       hipMemcpy( 
                   devDijkstra,
                   hostDevDijkstra,
                   pocetUzlu*sizeof(*devDijkstra), 
                   hipMemcpyHostToDevice 
                 )
    );

    delete [] hostDevDijkstra;
}

void inicializaceNtoN( unsigned **  graf, unsigned pocetUzlu, 
                       unsigned **& vzdalenostM,
                       unsigned **& devGraf, cDijkstra **& devDijkstra
                     ) {
    // inicializace matic vysledku -------------------------
    maticeInicializaceNaCPU( vzdalenostM, pocetUzlu );

    // zkopirovani grafu na GPU ----------------------------
    maticeInicializaceNaGPU( graf, pocetUzlu, devGraf );

    // inicializace objektu na GPU -------------------------
    dijkstraInicializaceNaGPU( devGraf, pocetUzlu, devDijkstra );

    // dalsi nastaveni pro GPU -----------------------------
#ifdef CACHE
    hipFuncSetCacheConfig(reinterpret_cast<const void*>( wrapperProGPU), hipFuncCachePreferL1 );
    //hipDeviceSetCacheConfig( hipFuncCachePreferL1 );
#endif // CACHE
}

void dijkstraObjektUklid( cDijkstra *& devDijkstra ) {
    unsigned * devVzdalenost;
    bool     * devUzavreny;
    // zkopirovani pointeru z [instance na GPU] ---------------------
    HANDLE_ERROR( 
            hipMemcpy( 
                &(devVzdalenost),
                &(devDijkstra->vzdalenost),
                sizeof(devVzdalenost),
                hipMemcpyDeviceToHost
                )
            );
    HANDLE_ERROR( 
            hipMemcpy( 
                &(devUzavreny),
                &(devDijkstra->uzavreny),
                sizeof(devUzavreny),
                hipMemcpyDeviceToHost
                )
            );
    // uvolneni pameti vnitrnich poli instance ----------------------
    HANDLE_ERROR( 
            hipFree( devVzdalenost )
            );
    HANDLE_ERROR( 
            hipFree( devUzavreny )
            );

    // uvolni objekt z pameti na GPU --------------------------------
    hipFree( devDijkstra );
    devDijkstra = NULL;
}

void dijkstraUklidNaGPU( cDijkstra **& devDijkstra, unsigned pocetUzlu ) {
    // zkopirovani pole [ukazatelu do device] ---------------------------
    cDijkstra ** hostDevDijkstra = new cDijkstra * [pocetUzlu];
    HANDLE_ERROR( 
       hipMemcpy( 
                   hostDevDijkstra,
                   devDijkstra,
                   pocetUzlu*sizeof(*devDijkstra), 
                   hipMemcpyDeviceToHost 
                 )
    );

    // uvolneni objektu z pameti GPU ---------------------------------
    for ( unsigned idUzlu = 0 ; idUzlu < pocetUzlu ; idUzlu++ ) {
        dijkstraObjektUklid( hostDevDijkstra[idUzlu] );
    }
    // uvolneni pole ukazatelu na objekty na GPU --------------------
    HANDLE_ERROR( 
            hipFree( devDijkstra )
            );
    devDijkstra = NULL;
    delete [] hostDevDijkstra;
}

void uklidNtoN( unsigned  ** vzdalenostM, 
                unsigned  ** devGraf,
                cDijkstra ** devDijkstra,
                unsigned     pocetUzlu 
              ) {
    maticeUklidNaCPU(   vzdalenostM, pocetUzlu );
    dijkstraUklidNaGPU( devDijkstra, pocetUzlu );
    maticeUklidNaGPU(       devGraf, pocetUzlu );
}

void zkopirujDataZGPU( unsigned ** vzdalenostM, cDijkstra ** devDijkstra, unsigned pocetUzlu ) {
    // zkopirovani pole [ukazatelu do device] ---------------------------
    cDijkstra ** hostDevDijkstra = new cDijkstra * [pocetUzlu];
    HANDLE_ERROR( 
            hipMemcpy( 
                hostDevDijkstra,
                devDijkstra,
                pocetUzlu*sizeof(*devDijkstra), 
                hipMemcpyDeviceToHost 
                )
            );

    unsigned * devHodnoty;
    for ( unsigned i = 0 ; i < pocetUzlu ; i++ ) {
        // zkopiruje hodnotu ukazatele z [tridy na device] ----------
        HANDLE_ERROR( 
                hipMemcpy( 
                    &(devHodnoty), 
                    &(hostDevDijkstra[i]->vzdalenost), 
                    //&(devDijkstra[i]->vzdalenost), 
                    sizeof(devHodnoty), 
                    hipMemcpyDeviceToHost
                    )
                );

        // zkopiruje data z device do matice vzdalenosti ------------
        HANDLE_ERROR( 
                hipMemcpy(
                    vzdalenostM[i],
                    devHodnoty,
                    pocetUzlu*sizeof(*devHodnoty),
                    hipMemcpyDeviceToHost 
                    )
                );
    }

    delete [] hostDevDijkstra;
}

__global__ void wrapperProGPU( cDijkstra ** devDijkstra, unsigned pocetUzlu, unsigned pocetVlakenVBloku ) {
    unsigned blok   =  blockIdx.x;
    unsigned vlakno = threadIdx.x;
    unsigned i      = pocetVlakenVBloku * blok + vlakno;

#ifdef DEBUG
    printf( "thread id = %d, b = %d, v = %d\n", i, blok, vlakno );
#endif // DEBUG

    if ( i < pocetUzlu ) {
        devDijkstra[i]->devInicializujHodnoty();
        devDijkstra[i]->devSpustVypocet();
    }
    return;
}

void dijkstraNtoN( unsigned ** graf, unsigned pocetUzlu, unsigned pocetWarpu ) {
    unsigned  ** devGraf;
    unsigned  ** vzdalenostM; 
    cDijkstra ** devDijkstra;
    // pocet vlaken v bloku -- minimalne pocet uzlu
    unsigned     vlaken = MIN( pocetWarpu * CUDA_WARP_VELIKOST, pocetUzlu );
    // horni cast pocetUzlu/vlaken
    unsigned     bloku  = ( pocetUzlu + vlaken - 1 ) / vlaken;

#ifdef MERENI
    // udalosti pro mereni casu vypoctu
    hipEvent_t udalosti[MERENI_POCET];
    float       tVypocet, tCelkem;

    mereniInicializace( udalosti, MERENI_POCET);
    mereniZaznam( udalosti[MERENI_START] );
#endif // MERENI

    // inicializace a kopirovani dat na GPU --------------------------
    inicializaceNtoN( graf, pocetUzlu, vzdalenostM, devGraf, devDijkstra );

#ifdef MERENI
    mereniZaznam( udalosti[MERENI_ZAPIS] );
#endif // MERENI

    // vypocet na GPU ------------------------------------------------
    wrapperProGPU <<< bloku, vlaken >>> ( devDijkstra, pocetUzlu, vlaken ) ;
    HANDLE_ERROR(   hipDeviceSynchronize( )        );

#ifdef MERENI
    mereniZaznam( udalosti[MERENI_VYPOCET] );
#endif // MERENI

    // kopirovani dat z GPU ------------------------------------------
    zkopirujDataZGPU( vzdalenostM, devDijkstra, pocetUzlu );

#ifdef MERENI
    mereniZaznam( udalosti[MERENI_KONEC] );
#endif // MERENI

#ifdef VYPIS
    // vypis vysledku ------------------------------------------------
    vypisGrafu( cout, vzdalenostM, pocetUzlu );
#endif // VYPIS

    // uvolneni pameti na CPU i GPU ----------------------------------
    uklidNtoN( vzdalenostM, devGraf, devDijkstra, pocetUzlu );

#ifdef MERENI
    mereniUplynulo( tVypocet, udalosti[MERENI_ZAPIS], udalosti[MERENI_VYPOCET] );
    mereniUplynulo(  tCelkem, udalosti[MERENI_START],   udalosti[MERENI_KONEC] );

    cerr << pocetUzlu << '	' << bloku   << '	' << vlaken << '	'
         << tVypocet  << '	' << tCelkem << endl;

    mereniUklid( udalosti, MERENI_POCET);

#endif // MERENI

}

