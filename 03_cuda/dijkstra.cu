#include "hip/hip_runtime.h"
/** dijkstra.cpp
 *
 * Autori:      Vojtech Myslivec <vojtech.myslivec@fit.cvut.cz>,  FIT CVUT v Praze
 *              Zdenek  Novy     <novyzde3@fit.cvut.cz>,          FIT CVUT v Praze
 *              
 * Datum:       unor-brezen 2015
 *
 * Popis:       Semestralni prace z predmetu MI-PAP:
 *              Hledani nejkratsich cest v grafu 
 *                 paralelni implementace na CUDA
 *                 algoritmus Dijkstra
 *
 *
 */

#include "dijkstra.h"
#include "cDijkstra.h"
#include "funkceSpolecne.h"

#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstring>
#include <omp.h>

using namespace std;

void dijkstraObjektInit( unsigned ** devGraf, unsigned pocetUzlu, unsigned idUzlu, cDijkstra *& devDijkstra ) {
    // vytvori objekt na host, aby ho zkopiroval na device ----------
    cDijkstra * hostDijkstra = new cDijkstra( pocetUzlu, idUzlu ); 

    hipMalloc( &devDijkstra, sizeof(*devDijkstra) );
    hipMemcpy( devDijkstra, hostDijkstra, sizeof(*devDijkstra), hipMemcpyHostToDevice );

    delete hostDijkstra;

    // alokace vnitrnich poli ---------------------------------------
    unsigned * devVzdalenost;
    bool     * devUzavreny;
    HANDLE_ERROR( 
       hipMalloc( 
                   &devVzdalenost,
                   pocetUzlu*sizeof(*devVzdalenost)
                 )
    );
    HANDLE_ERROR( 
       hipMalloc( 
                   &devUzavreny,
                   pocetUzlu*sizeof(*devUzavreny)
                 )
    );

    // zkopirovani pointeru do instance -----------------------------
    HANDLE_ERROR( 
       hipMemcpy( 
                   &(devDijkstra->vzdalenost),
                   &(devVzdalenost),
                   sizeof(devVzdalenost),
                   hipMemcpyHostToDevice
                 )
    );
    HANDLE_ERROR( 
       hipMemcpy( 
                   &(devDijkstra->uzavreny),
                   &(devUzavreny),
                   sizeof(devUzavreny),
                   hipMemcpyHostToDevice
                 )
    );

    // zkopruje ukazatel na pseudo-staticky graf --------------------
    HANDLE_ERROR( 
       hipMemcpy( 
                   &(devDijkstra->graf),
                   &(devGraf),
                   sizeof(devGraf),
                   hipMemcpyHostToDevice
                 )
    );
}

void dijkstraInicializaceNaGPU( unsigned ** devGraf, unsigned pocetUzlu, cDijkstra **& devDijkstra ) {
    cDijkstra ** hostDevDijkstra = new cDijkstra * [pocetUzlu];

    // alokace objektu na GPU ---------------------------------------
    // TODO smazat? 
    // devDijkstra = new cDijkstra * [pocetUzlu];
    for ( unsigned idUzlu = 0 ; idUzlu < pocetUzlu ; idUzlu++ ) {
        dijkstraObjektInit( devGraf, pocetUzlu, idUzlu, hostDevDijkstra[idUzlu] );
    }
    // alokace a zkopirovani pole ukazatelu na GPU ------------------
    HANDLE_ERROR( 
       hipMalloc( 
                   &devDijkstra,
                   pocetUzlu*sizeof(*devDijkstra)
                 )
    );
    HANDLE_ERROR( 
       hipMemcpy( 
                   devDijkstra,
                   hostDevDijkstra,
                   pocetUzlu*sizeof(*devDijkstra), 
                   hipMemcpyHostToDevice 
                 )
    );

    delete [] hostDevDijkstra;

}

void grafInicializaceNaGPU( unsigned ** graf, unsigned pocetUzlu, unsigned **& devGraf ) {
    // alokace matice -- pole sloupcu ---------------------
    HANDLE_ERROR( 
            hipMalloc( 
                &devGraf,
                pocetUzlu*sizeof(*devGraf)
                )
            ); 

    // v cyklu se alokuji a kopiruji data z grafu na GPU
    unsigned * devHodnoty;
    for ( unsigned i = 0 ; i < pocetUzlu ; i++ ) {
        // alokace jednoho radku matice -------------------
        HANDLE_ERROR( 
                hipMalloc( 
                    &devHodnoty,
                    pocetUzlu*sizeof(*devHodnoty)
                    )
                );
        // kopirovani jednoho radku matice ----------------
        HANDLE_ERROR( 
                hipMemcpy( 
                    devHodnoty,
                    graf[i],
                    pocetUzlu*sizeof(*devHodnoty), 
                    hipMemcpyHostToDevice 
                    )
                );
        // zkopirovani pointeru na radek do pole sloupcu --
        HANDLE_ERROR( 
                hipMemcpy( 
                    &(devGraf[i]),
                    &(devHodnoty),
                    sizeof(devHodnoty),
                    hipMemcpyHostToDevice
                    )
                );
    }
}

void inicializaceNtoN( unsigned ** graf,     unsigned pocetUzlu, 
                       unsigned **& vzdalenostM,
                       unsigned **& devGraf, cDijkstra **& devDijkstra
                     ) {
    // inicializace matic vysledku -------------------------
    vzdalenostM = new unsigned*[pocetUzlu];
    for ( unsigned i = 0; i < pocetUzlu; i++ ) {
        vzdalenostM[i] = new unsigned[pocetUzlu];
        for ( unsigned j = 0; j < pocetUzlu; j++ ) {
            vzdalenostM[i][j] = DIJKSTRA_NEKONECNO;
        }
    }

    // pseudo-staticka inicializace -----------------------
    grafInicializaceNaGPU( graf, pocetUzlu, devGraf );

    // inicializace objektu na GPU ------------------------
    dijkstraInicializaceNaGPU( devGraf, pocetUzlu, devDijkstra );
}

void uklidUkazatelu( unsigned **& dveDimenze, unsigned rozmer ) {
    if ( dveDimenze != NULL ) {
        for ( unsigned i = 0; i < rozmer; i++ ) {
            if ( dveDimenze[i] != NULL ) {
                delete [] dveDimenze[i];
                dveDimenze[i] = NULL;
            }
        }
        delete [] dveDimenze;
        dveDimenze = NULL;
    }
}

void dijkstraObjektUklid( cDijkstra *& devDijkstra ) {
    unsigned * devVzdalenost;
    bool     * devUzavreny;
    // zkopirovani pointeru z [instance na GPU] ---------------------
    HANDLE_ERROR( 
            hipMemcpy( 
                &(devVzdalenost),
                &(devDijkstra->vzdalenost),
                sizeof(devVzdalenost),
                hipMemcpyDeviceToHost
                )
            );
    HANDLE_ERROR( 
            hipMemcpy( 
                &(devUzavreny),
                &(devDijkstra->uzavreny),
                sizeof(devUzavreny),
                hipMemcpyDeviceToHost
                )
            );
    // uvolneni pameti vnitrnich poli instance ----------------------
    HANDLE_ERROR( 
            hipFree( devVzdalenost )
            );
    HANDLE_ERROR( 
            hipFree( devUzavreny )
            );

    // uvolni objekt z pameti na GPU --------------------------------
    hipFree( devDijkstra );
    devDijkstra = NULL;

}

void dijkstraUklidNaGPU( cDijkstra **& devDijkstra, unsigned pocetUzlu ) {
    // uvolneni objektu z pameti GPU ---------------------------------
    for ( unsigned idUzlu = 0 ; idUzlu < pocetUzlu ; idUzlu++ ) {
        dijkstraObjektUklid( devDijkstra[idUzlu] );
    }
    // uvolneni pole ukazatelu na objekty na GPU --------------------
    HANDLE_ERROR( 
            hipFree( devDijkstra )
            );
    devDijkstra = NULL;
}

void grafUklidNaGPU( unsigned **& devGraf, unsigned pocetUzlu ) {
    // v cyklu se kopiruji hodnoty ukazatelu na radek a ty se smazou
    unsigned * devHodnoty;
    for ( unsigned i = 0 ; i < pocetUzlu ; i++ ) {
        // zkopirovani ukazatele z [pole sloupcu na device] --
        HANDLE_ERROR( 
                hipMemcpy( 
                    &(devHodnoty),
                    &(devGraf[i]),
                    sizeof(devHodnoty),
                    hipMemcpyDeviceToHost
                    )
                );
        // uvolneni pameti radku matice -------------------
        HANDLE_ERROR( 
                hipFree( devHodnoty )
                );
    }
    // uvolneni pameti ukazatele na radky matice ----------
    HANDLE_ERROR( 
       hipFree( devGraf )
    ); 
    devGraf = NULL;
}

void uklidNtoN( unsigned  ** vzdalenostM, 
                unsigned  ** devGraf,
                cDijkstra ** devDijkstra,
                unsigned     pocetUzlu 
              ) {
    uklidUkazatelu(     vzdalenostM, pocetUzlu );
    dijkstraUklidNaGPU( devDijkstra, pocetUzlu );
    grafUklidNaGPU(         devGraf, pocetUzlu );
}

void zkopirujDataZGPU( unsigned ** vzdalenostM, cDijkstra ** devDijkstra, unsigned pocetUzlu ) {
//     // zkopirovani pole [ukazatelu do device] ---------------------------
//     cDijkstra ** hostDevDijkstra = new cDijkstra* [pocetUzlu];
//     HANDLE_ERROR( 
//        hipMemcpy( 
//                    hostDevDijkstra,
//                    devDijkstra,
//                    pocetUzlu*sizeof(*devDijkstra), 
//                    hipMemcpyDeviceToHost 
//                  )
//     );

    unsigned * devHodnoty;
    for ( unsigned i = 0 ; i < pocetUzlu ; i++ ) {
        // zkopiruje hodnotu ukazatele z [tridy na device] ----------
        HANDLE_ERROR( 
            hipMemcpy( 
                        &(devHodnoty), 
                        //&(hostDevDijkstra[i]->vzdalenosti), 
                        &(devDijkstra[i]->vzdalenost), 
                        sizeof(devHodnoty), 
                        hipMemcpyDeviceToHost
                      )
        );

        // zkopiruje data z device do matice vzdalenosti ------------
        HANDLE_ERROR( 
           hipMemcpy(
                       vzdalenostM[i],
                       devHodnoty,
                       pocetUzlu*sizeof(*devHodnoty),
                       hipMemcpyDeviceToHost 
                     )
        );
    }
    
//     delete [] hostDevDijkstra;
}

__global__ void wrapperProGPU( cDijkstra ** devDijkstra, unsigned pocetUzlu ) {
    for ( unsigned i = 0 ; i < pocetUzlu ; i++ ) {
        devDijkstra[i]->devInicializujHodnoty();
        devDijkstra[i]->devSpustVypocet();
    }
    return;
}

bool dijkstraNtoN( unsigned ** graf, unsigned pocetUzlu, unsigned pocetVlaken ) {
    unsigned  ** devGraf;
    unsigned  ** vzdalenostM; 
    cDijkstra ** devDijkstra;

    inicializaceNtoN( graf, pocetUzlu, vzdalenostM, devGraf, devDijkstra );

    wrapperProGPU<<<1,1>>>( devDijkstra, pocetUzlu ) ;
    hipDeviceSynchronize();

    zkopirujDataZGPU( vzdalenostM, devDijkstra, pocetUzlu );

    vypisVysledekMaticove( vzdalenostM, pocetUzlu );

    uklidNtoN( vzdalenostM, devGraf, devDijkstra, pocetUzlu );

    return true;
}

void vypisVysledekMaticove( unsigned ** vzdalenosti, unsigned pocetUzlu ) {
    cout << "Vzdalenosti:" << endl;
    vypisGrafu( cout, vzdalenosti, pocetUzlu );
}

