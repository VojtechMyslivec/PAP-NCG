#include "hip/hip_runtime.h"
/** floydWarshall.cu
 *
 * Autori:      Vojtech Myslivec <vojtech.myslivec@fit.cvut.cz>,  FIT CVUT v Praze
 *              Zdenek  Novy     <novyzde3@fit.cvut.cz>,          FIT CVUT v Praze
 *              
 * Datum:       unor-kveten 2015
 *
 * Popis:       Semestralni prace z predmetu MI-PAP:
 *              Hledani nejkratsich cest v grafu 
 *                 paralelni implementace na CUDA
 *                 funkce pro algoritmus Floyd-Warshall
 *
 *
 */

#include "floydWarshall.cuh"
#ifdef DEBUG
   #include <stdio.h>
#endif // DEBUG

// funkce pro inicializovani veskerych promennych potrebnych behem vypoctu 
void inicializace( unsigned ** graf, unsigned pocetUzlu, unsigned **& hostDelka, unsigned **& devDelka ) {
    // alokovani pameti pro vysledek -----------------------
    maticeInicializaceNaCPU( hostDelka, pocetUzlu );

    // alokovani pameti na GPU -----------------------------
    maticeInicializaceNaGPU(   graf, pocetUzlu, devDelka );

    // dalsi nastaveni pro GPU -----------------------------
#ifdef CACHE
    hipDeviceSetCacheConfig( hipFuncCachePreferL1 );
#endif // CACHE
}

// funkce, ktera zajisti uklizeni alokovanych promennych
void uklid( unsigned pocetUzlu, unsigned **& hostDelka, unsigned **& devDelka ) {
    maticeUklidNaCPU(       hostDelka, pocetUzlu );
    maticeUklidNaGPU(  devDelka, pocetUzlu );

    hostDelka    = NULL;
    devDelka = NULL;
}

void zkopirujDataZGPU( unsigned ** hostDelka, unsigned ** devDelka, unsigned pocetUzlu ) {
    // zkopirovani pole [ukazatelu do device] ---------------------------
    unsigned ** hostDevVzdalenost = new unsigned * [pocetUzlu];
    HANDLE_ERROR( 
            hipMemcpy( 
                hostDevVzdalenost,
                devDelka,
                pocetUzlu*sizeof(*devDelka), 
                hipMemcpyDeviceToHost 
                )
            );

    for ( unsigned i = 0 ; i < pocetUzlu ; i++ ) {
        // zkopiruje data z device do matice vzdalenosti ------------
        HANDLE_ERROR( 
                hipMemcpy(
                    hostDelka[i],
                    hostDevVzdalenost[i],
                    pocetUzlu*sizeof(*hostDevVzdalenost[i]),
                    hipMemcpyDeviceToHost 
                    )
                );
    }

    delete [] hostDevVzdalenost;
}

// vzdy se spousti 32 bloku x 32 vlaken -- DLAZDICE_VELIKOST x DLAZDICE_VELIKOST 
__global__ void kernelProNezavisleDlazdice( unsigned ** devDelka, unsigned pocetUzlu, unsigned dlazdice, unsigned krok ) {
    const unsigned radek   =  blockIdx.x + dlazdice * DLAZDICE_VELIKOST;
    const unsigned sloupec = threadIdx.x + dlazdice * DLAZDICE_VELIKOST;
#ifdef DEBUG
    const unsigned blok   =  blockIdx.x;
    const unsigned vlakno = threadIdx.x;
    const unsigned id     = DLAZDICE_VELIKOST * blok + vlakno;
    printf( "  - Kernel 1: vlakno id = %d, b = %d, v = %d, M[ %d , %d ]\n", id, blok, vlakno, radek, sloupec );
#endif // DEBUG

    if ( radek < pocetUzlu && sloupec < pocetUzlu ) {
        devDelka[radek][sloupec] = MIN(  devDelka[radek][sloupec],  devDelka[radek][krok] + devDelka[krok][sloupec]  );
    }
}

// vzdy se spousti 32 bloku x 32 vlaken -- DLAZDICE_VELIKOST x DLAZDICE_VELIKOST 
__global__ void kernelProJednoZavisleDlazdice( unsigned ** devDelka, unsigned pocetUzlu, unsigned dlazdiceRadek, unsigned dlazdiceSloupec, unsigned krok ) {
    const unsigned radek   =  blockIdx.x + dlazdiceRadek   * DLAZDICE_VELIKOST;
    const unsigned sloupec = threadIdx.x + dlazdiceSloupec * DLAZDICE_VELIKOST;
#ifdef DEBUG
    const unsigned blok   =  blockIdx.x;
    const unsigned vlakno = threadIdx.x;
    const unsigned id     = DLAZDICE_VELIKOST * blok + vlakno;
    printf( "  - Kernel 2: vlakno id = %d, b = %d, v = %d, M[ %d , %d ]\n", id, blok, vlakno, radek, sloupec );
#endif // DEBUG

    if ( radek < pocetUzlu && sloupec < pocetUzlu ) {
        devDelka[radek][sloupec] = MIN(  devDelka[radek][sloupec],  devDelka[radek][krok] + devDelka[krok][sloupec]  );
    }
}

__global__ void kernelProDvouZavisleDlazdice( unsigned ** devDelka, unsigned pocetUzlu, unsigned dlazdiceRadek, unsigned dlazdiceSloupec, unsigned krok ) {
    const unsigned radek   =  blockIdx.x + dlazdiceRadek   * DLAZDICE_VELIKOST;
    const unsigned sloupec = threadIdx.x + dlazdiceSloupec * DLAZDICE_VELIKOST;
#ifdef DEBUG
    const unsigned blok   =  blockIdx.x;
    const unsigned vlakno = threadIdx.x;
    const unsigned id     = DLAZDICE_VELIKOST * blok + vlakno;
    printf( "  - Kernel 2: vlakno id = %d, b = %d, v = %d, M[ %d , %d ]\n", id, blok, vlakno, radek, sloupec );
#endif // DEBUG

    if ( radek < pocetUzlu && sloupec < pocetUzlu ) {
        devDelka[radek][sloupec] = MIN(  devDelka[radek][sloupec],  devDelka[radek][krok] + devDelka[krok][sloupec]  );
    }
}

// realizuje samotny (paralelni) vypocet algoritmu Floyd-Warshalla O( n^3 / p ) 
void spustVypocet( unsigned ** devDelka, unsigned pocetUzlu, unsigned pocetWarpu ) {
    const unsigned s = DLAZDICE_VELIKOST;
    // horni cast pocetUzlu / s
    const unsigned pocetDlazdic = ( pocetUzlu + s - 1 ) / s; 

//    unsigned vlakenMin    = pocetUzlu * pocetUzlu;
//    // pocet vlaken v bloku -- minimalne pocet uzlu ^ 2
//    unsigned vlakenVBloku = MIN( pocetWarpu * CUDA_WARP_VELIKOST, vlakenMin );
//    // horni cast pocetUzlu/vlakenVBloku
//    unsigned bloku        = ( vlakenMin + vlakenVBloku - 1 ) / vlakenVBloku;

    for ( unsigned b = 0 ; b < pocetDlazdic ; b++ ) {
#ifdef DEBUG
        printf( "b = %d\n", b );
#endif // DEBUG
        // nezavisle dlazdice -- na hl. diagonale dlazdickovane matice ---------
        for ( unsigned k = b*s ; k < (b+1)*s ; k++ ) {
            if ( k >= pocetUzlu ) break;            // pokud je uz mimo, konci
            kernelProNezavisleDlazdice <<< s, s >>> ( devDelka, pocetUzlu, b, k );
            HANDLE_ERROR(   hipDeviceSynchronize( ) );
        }

        // jedno-zavisle dlazdice ----------------------------------------------
        // ve stejnem radku
        for ( unsigned ib = 0 ; ib < pocetDlazdic ; ib++ ) {
            if ( ib == b ) continue;    // pokud uz danou dlazdici spocital, preskoci
            for ( unsigned k = b*s ; k < (b+1)*s ; k++ ) {
                if ( k >= pocetUzlu ) break;            // pokud je uz mimo, konci
                kernelProJednoZavisleDlazdice <<< s, s >>> ( devDelka, pocetUzlu, b, ib, k );
                HANDLE_ERROR(   hipDeviceSynchronize( ) );
            }
        }
        // ve stejnem sloupci
        for ( unsigned jb = 0 ; jb < pocetDlazdic ; jb++ ) {
            if ( jb == b ) continue;    // pokud uz danou dlazdici spocital, preskoci
            for ( unsigned k = b*s ; k < (b+1)*s ; k++ ) {
                if ( k >= pocetUzlu ) break;            // pokud je uz mimo, konci
                kernelProJednoZavisleDlazdice <<< s, s >>> ( devDelka, pocetUzlu, jb, b, k );
                HANDLE_ERROR(   hipDeviceSynchronize( ) );
            }
        }

        // dvou-zavisle dlazdice -- zbytek -------------------------------------
        for ( unsigned ib = 0 ; ib < pocetDlazdic ; ib++ ) {
            if ( ib == b ) continue;        // pokud uz danou dlazdici spocital, preskoci
            for ( unsigned jb = 0 ; jb < pocetDlazdic ; jb++ ) {
                if ( jb == b ) continue;    // pokud uz danou dlazdici spocital, preskoci
                for ( unsigned k = b*s ; k < (b+1)*s ; k++ ) {
                    if ( k >= pocetUzlu ) break;    // pokud je uz mimo, konci
                    kernelProDvouZavisleDlazdice <<< s, s >>> ( devDelka, pocetUzlu, jb, ib, k );
                }
            }
        }

#ifdef DEBUG
        printf( "\n" );
#endif // DEBUG
    }
}

void floydWarshall( unsigned ** graf, unsigned pocetUzlu, unsigned pocetWarpu ) {
    unsigned ** devDelka  = NULL;
    unsigned ** hostDelka = NULL;
    
#ifdef MERENI
    // udalosti pro mereni casu vypoctu
    hipEvent_t udalosti[MERENI_POCET];
    float       tVypocet, tCelkem;

    mereniInicializace( udalosti, MERENI_POCET);
    mereniZaznam( udalosti[MERENI_START] );
#endif // MERENI

    // inicializace a kopirovani dat na GPU --------------------------
    inicializace( graf, pocetUzlu, hostDelka, devDelka );

#ifdef MERENI
    mereniZaznam( udalosti[MERENI_ZAPIS] );
#endif // MERENI

    // vypocet na GPU ------------------------------------------------
    spustVypocet( devDelka, pocetUzlu, pocetWarpu );
    HANDLE_ERROR(   hipDeviceSynchronize( )        );

#ifdef MERENI
    mereniZaznam( udalosti[MERENI_VYPOCET] );
#endif // MERENI

    // kopirovani dat z GPU ------------------------------------------
    zkopirujDataZGPU( hostDelka, devDelka, pocetUzlu );

#ifdef MERENI
    mereniZaznam( udalosti[MERENI_KONEC] );
#endif // MERENI

#ifdef VYPIS
    // vypis vysledku ------------------------------------------------
    vypisGrafu( cout, hostDelka, pocetUzlu );
#endif // VYPIS

    // uvolneni pameti na CPU i GPU ----------------------------------
    uklid( pocetUzlu, hostDelka, devDelka );

#ifdef MERENI
    mereniUplynulo( tVypocet, udalosti[MERENI_ZAPIS], udalosti[MERENI_VYPOCET] );
    mereniUplynulo(  tCelkem, udalosti[MERENI_START],   udalosti[MERENI_KONEC] );

    cerr << pocetUzlu << '	' //<< bloku   << '	' << vlakenVBloku << '	'
         << tVypocet  << '	' << tCelkem << endl;

    mereniUklid( udalosti, MERENI_POCET);
#endif // MERENI

}

